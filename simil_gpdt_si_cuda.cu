/**
 * Programma che simula il comportamento del gpdt per 
 * la risoluzione di un kernel di una serie di
 * valori di dimensione variabile utilizzando la 
 * tecnologia cuda.
 * compilare con:
 * nvcc -o simil_gpdt_si_cuda simil_gpdt_si_cuda.cu
 * lanciare con:
 * ./simil_gpdt_si_cuda [numero vettori] [numero componenti] [numero di righe da calcolare] [tipo di kernel] [grado(int)/sigma(float)]
 **/

#include <iostream>
#include <ctime>
#include <cstdlib>
#include <cstdio>
#include <math.h>
#include <hip/hip_runtime.h>
using namespace std;

/**
 * Funzione che riempie i vettori con numeri
 * casuali compresi tra 0 e 99.
 **/ 
void riempi_vettori(float *vettori, int Nr_vet_elem, int Nr_vet_comp)
{
	for (int i = 0; i < Nr_vet_elem; i++)
		for(int j = 0; j < Nr_vet_comp; j++)
			vettori[i * Nr_vet_comp + j] = i * 2 + j; //j % 4; //
}

/**
 * Funzione che crea dei vettori contenente i valori significativi su cui 
 * calcolare la norma 2 al quadrato.
 **/
void crea_vettori_termini_noti(int *vettori, int Nr_vet_elem, int Nr_vet_comp)
{
	for (int i = 0; i < Nr_vet_elem; i++)
		for(int j = 0; j < Nr_vet_comp; j++)
			vettori[i * Nr_vet_comp + j] = (j+1)*3;
}

/**
 * Funzione che crea un vettore contenente il numero di valori significative.
 **/
void crea_vettori_posizioni(int *vettore, int Nr_vet_elem, int numero_val)
{
	for (int i = 0; i < Nr_vet_elem; i++)
		vettore[i] = numero_val;
}

/**
 * kernel per il calcolo delle norme al quadrato dei vettori.
 **/
__global__ void Kernel_norme(float *Vd, float *Nd, int *Vp, int *Vnp, int N, int C, int nr_max_val)
{
	long int x = threadIdx.x + blockIdx.x * blockDim.x;
	
	int pos;
	
	if(x < N)
	{
		float norma = 0;
		
		int Nr_val = Vnp[x];
		
		for(int i = 0; i < Nr_val; i++)
		{
			pos = Vp[x * nr_max_val + i];
			norma = norma + (Vd[x * C + pos] * Vd[x * C + pos]);
		}
		
		Nd[x] = norma;
	}
	
} 

/**
 * Kernel per il calcolo del del guassiano, basato sul metodo utilizzato nel gpdt, 
 * modificato per l'utilizzo con la tecnologia CUDA.
 **/
__global__ void Kernel_gaus(float *Vd, float *Ris, float *Nd, int N, int C, int dim_indici, int *ind, float sigma, int *Vp, int *Vnp, int nr_max_val)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int j;
	int pos;
	int tmp_ind;
	float gaus;
	
	for ( ; x < N ; x+=blockDim.x * gridDim.x)
	{
	    for( ; y < dim_indici; y+=blockDim.y * gridDim.y)
	    {
		      tmp_ind = ind[y];
		    
		      gaus = 0.0;
		      
		      int Nr_val = Vnp[x];
		      
		      for(j = 0; j < Nr_val; j++)
		      {
			      pos = Vp[x * nr_max_val + j];
			      gaus = gaus + (Vd[x * C + pos] * Vd[tmp_ind * C + pos]);
		      }
		      
		      gaus = - 2.0*gaus +Nd[x] + Nd[tmp_ind];
		      gaus = (exp(-gaus*sigma));

		      //Ris[x * dim_indici + y] = gaus;
		      Ris[y * N + x] = gaus;
	    }
	}
}

/**
 * Kernel per il calcolo del kernel lineare 
 * modificato per l'utilizzo con la tecnologia CUDA.
 **/
__global__ void Kernel_lineare(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int j;
	int pos;
	int tmp_ind;
	float lin;
	
	for ( ; x < N ; x+=blockDim.x * gridDim.x)
	{
	    for( ; y < dim_indici; y+=blockDim.y * gridDim.y)
	    {
		      tmp_ind = ind[y];
		    
		      lin = 0.0;
		      
		      int Nr_val = Vnp[x];
		      
		      for(j = 0; j < Nr_val; j++)
		      {
			      pos = Vp[x * nr_max_val + j];
			      lin =  lin + (Vd[x * C + pos] * Vd[tmp_ind * C + pos]);
		      }

		      //Ris[x * dim_indici + y] = lin;
		      Ris[y * N + x ] = lin;
	    }
	}
}


/**
 * Kernel per il calcolo del kernel lineare 
 * modificato per l'utilizzo con la tecnologia CUDA.
 **/
__global__ void Kernel_polimoniale(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val, int s)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	int j;
	int pos;
	int tmp_ind;
	float pol;
	float tmp;
	
	for ( ; x < N ; x+=blockDim.x * gridDim.x)
	{
	    for( ; y < dim_indici; y+=blockDim.y * gridDim.y)
	    {
		      tmp_ind = ind[y];
			  
			  tmp = 1.0;
			  
		      pol = 0.0;
		      
		      int Nr_val = Vnp[x];
		      
		      for(j = 0; j < Nr_val; j++)
		      {
			      pos = Vp[x * nr_max_val + j];
			      pol = pol + (Vd[x * C + pos] * Vd[tmp_ind * C + pos]);
		      }
		      
		      pol = pol + 1;
		      
		      for(j = 0; j < s; j++)
		      {
				tmp = tmp * pol;
			  }

		      //Ris[x * dim_indici + y] = tmp;
		      Ris[y * N + x ] = tmp;
	    }
	}
}



int main(int argc, char** argv)

{	
	/**
	 * Variabile contenente la percentuale dei valori significativi 
	 * all'interno dei vettori.
	 **/
	float perc_val_noti= 1.0 - 0.82;
	
	/**
	 * Matrice contenente i vettori da cui calcolare la differeza 
	 * per calcolarne le norme.
	 * Il numero di vettori e la dimensione degli stessi viene definita
	 * dall'utente.
	 **/
	float *vettori;
	
	/**
	 * Matrice contenente le posizioni all'interno del vettore contenente
	 * le posizioni dei valori significativi.
	 **/
	 int *vettore_posizioni;
	
	/**
	 * Vettore contenente il numero di valori non nulli nel vettore.
	 **/
	int *vett_numero_posizioni;
	
	/**
	 * vettore contenente le norme 2 al quadreato dei vettori.
	 **/
	float *vett_norme;
	
	/**
	 * Matrice contenente i risultati.
	 **/
	float *risultati;
	
	/**
	 * 
	 **/
	int *indici; 
	
	/**
	 * Tempo impiegato per il calcolo
	 **/
	float elapsedTime;
	hipEvent_t start, stop;
	
	//nr di vettori e di elementi.
	int Nr_vet_elem = atoi(argv[1]);
	int Nr_vet_comp = atoi(argv[2]);
	
	//Numero di righe da calcolare.
	int Nr_righe = atoi(argv[3]);
	
	/**
	 * Numero per la selezione del kernel.
	 * 1 = kernel lineare.
	 * 2 = kernel polimoniale.
	 * 3 = kernel gaussiano.
	 **/
	int sel_kernel = atoi(argv[4]);
	
	/**
	 * Sigma della funzione gaussiana.
	 **/
	//float sigma = atoi(argv[5]);
	//sigma = (1.0/(2.0*sigma*sigma));
	
	//Copia per il device.
	float *Vd;
	int *Vp;
	int *Vnp;
	float *Nd;
	float *Ris;
	int *ind;
	
	//Variabili per il controllo della memoria disponibile.
	size_t free_byte;
	size_t total_byte;
	
	/**
	 * Variabile contenente il numero dei valori significativi.
	 **/
	int numero_val_significativi = Nr_vet_comp * perc_val_noti;
	
	//Spazio necessario per l'allocazione dei vettori.
	int tot_vett_size = Nr_vet_elem * Nr_vet_comp * sizeof(float);
	//Spazio necessario per l'allocazione della Matrice dei risultati.
	int norme_size = Nr_vet_elem * sizeof(float);
	//Spazio necessario per l'allocazione della Matrice delle posizioni.
	int vett_pos_size = Nr_vet_elem * numero_val_significativi * sizeof(int);
	//Spazio necessario per l'allocazione del vettore con il numero dei valori significativi.
	int vett_nrpos_size = Nr_vet_elem * sizeof(int);
	//Spazio necessario per l'allocazione di una colonna.
	int col_size = Nr_vet_elem * sizeof(float);
	
	
	
	//Allocazione.
	vettori = (float*)malloc(tot_vett_size);
	vett_norme = (float*)malloc(norme_size);
	vettore_posizioni = (int*)malloc(vett_pos_size);
	vett_numero_posizioni = (int*)malloc(vett_nrpos_size);
	
	
	
	//Allocazione nel device.
	hipMalloc((void **)&Vd, tot_vett_size);
	hipMalloc((void **)&Nd, norme_size);
	hipMalloc((void **)&Vp, vett_pos_size);
	hipMalloc((void **)&Vnp, vett_nrpos_size);
	
	srand(time(0));
	
	//Riempimento dei vettori.
	riempi_vettori(vettori, Nr_vet_elem, Nr_vet_comp);
	//Riempimento dei vettori delle posizioni.
	crea_vettori_termini_noti(vettore_posizioni, Nr_vet_elem, numero_val_significativi);
	//Riempimento del vettore contenente il numero dei valori significativi.
	crea_vettori_posizioni(vett_numero_posizioni, Nr_vet_elem, numero_val_significativi);
		
	//trasferimento dei vettori nel device.
	hipMemcpy(Vd, vettori, tot_vett_size, hipMemcpyHostToDevice);
	//trasferimento dei vettori delle posizioni nel device.
	hipMemcpy(Vp, vettore_posizioni, vett_pos_size, hipMemcpyHostToDevice);
	//trasferimento del vettore conentente il numero di valori all'interno di ogni singolo vettore.
	hipMemcpy(Vnp, vett_numero_posizioni, vett_nrpos_size, hipMemcpyHostToDevice);
	
	hipMemGetInfo( &free_byte, &total_byte );
	
	int col_ospitabili_mem = (free_byte*0.7)/col_size;
	
	int contatore = 0;
	
	/**
	 * Valori impostati per ottimizzare il funzionamento del device.
	 * Questi valori sono basati sull'utilizzo di una Nvidia 230m.
	 **/
	int dimXX =4;
	int dimYY =128;
	
	/**
	 * Numero di colonne ospitabili calcolabili dal kernel contemporaneamente.
	 * Purtroppo a causa del fatto che il kernel CUDA fallisca in automatico
	 * se impiega più di 5 secondi per il calcolo, è necessario inserire un 
	 * limitatore per il calcolo.
	 * Questo valore è basato sull'utilizzo di una Nvidia 230m.
	 **/
	 
	
	int col_ospitabili = 200;
	
	if (col_ospitabili > Nr_righe)
	{
		col_ospitabili = Nr_righe;
	} 
	
	if (col_ospitabili > col_ospitabili_mem)
	{
		col_ospitabili = col_ospitabili_mem;
	}
	
	int numero_cicli = Nr_righe/col_ospitabili;
	cout<<"Numero cicli necessari: "<<numero_cicli<<endl;
	
	int risultati_size = Nr_righe * Nr_vet_elem * sizeof(float);
	int indici_size = col_ospitabili * sizeof(int);
	
	int risultati_part_size = col_ospitabili * Nr_vet_elem * sizeof(float);
	
	risultati = (float*)malloc(risultati_size);
	indici = (int*)malloc(indici_size);
	
	int numSMs;
	hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
	
	hipMalloc((void **)&Ris, risultati_part_size);
	hipMalloc((void **)&ind, indici_size);
	
	dim3 blockGridRows;
	
	blockGridRows.x=Nr_vet_elem/dimXX + (Nr_vet_elem%dimXX== 0?0:1);;
	blockGridRows.y=col_ospitabili/dimYY + (col_ospitabili%dimYY== 0?0:1);
	
	dim3 threadBlockRows;
	threadBlockRows.x=dimXX;
	threadBlockRows.y=dimYY;
	
	cout<<"Memoria allocata, griglie definite:"<<endl;
	cout<<"blockGridRows.x: "<<blockGridRows.x<<endl;
	cout<<"blockGridRows.y: "<<blockGridRows.y<<endl;
	cout<<"threadBlockRows.x: "<<threadBlockRows.x<<endl;
	cout<<"threadBlockRows.y: "<<threadBlockRows.y<<endl;
	
	int cicle_dim = col_ospitabili * Nr_vet_elem * sizeof(float);
	
	dim3 blockGridRowsn(Nr_vet_elem, 1);
	
	int resto;
	
	hipEventCreate(&start);
	hipEventRecord(start,0);
	
	switch(sel_kernel){
			case(1):{
				//cout<<"Kernel Lineare\n";
							
				for(int i = 0; i < numero_cicli; i++)
				{
					for (int kk = 0; kk < col_ospitabili; kk++)
					{
						indici[kk] = contatore + kk;
					}
					
					hipMemcpy(ind, indici, indici_size, hipMemcpyHostToDevice);
					
					//__global__ void Kernel_lineare(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val)
					Kernel_lineare<<< blockGridRows, threadBlockRows>>>(Vd, Ris, Nr_vet_elem, Nr_vet_comp, col_ospitabili, ind, Vp, Vnp, numero_val_significativi);
					
					hipMemcpy(risultati+(i*col_ospitabili*Nr_vet_elem), Ris, cicle_dim, hipMemcpyDeviceToHost);
					
					contatore = contatore + col_ospitabili;
				}
				
				resto = Nr_righe - contatore;
				
				if (resto > 0)
				{
					for (int kk = 0; kk < resto; kk++)
					{
						indici[kk] = contatore + kk;
					}
					
					hipMemcpy(ind, indici, indici_size, hipMemcpyHostToDevice);
					
					Kernel_lineare<<< blockGridRows, threadBlockRows>>>(Vd, Ris, Nr_vet_elem, Nr_vet_comp, resto, ind, Vp, Vnp, numero_val_significativi);
					
					hipMemcpy(risultati+(numero_cicli)*(col_ospitabili*Nr_vet_elem), Ris, resto * Nr_vet_elem * sizeof(float), hipMemcpyDeviceToHost);
					
				}
				
				break;
			}
			
			case(2):{
				//cout<<"Kernel Polimoniale\n";
				/**
				 * Grado del kernel.
				 **/
				int grado = atoi(argv[5]);
				
				for(int i = 0; i < numero_cicli; i++)
				{
					for (int kk = 0; kk < col_ospitabili; kk++)
					{
						indici[kk] = contatore + kk;
					}
					
					hipMemcpy(ind, indici, indici_size, hipMemcpyHostToDevice);
					
					//__global__ void Kernel_polimoniale(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val, int s)
					Kernel_polimoniale<<< blockGridRows, threadBlockRows>>>(Vd, Ris, Nr_vet_elem, Nr_vet_comp, col_ospitabili, ind, Vp, Vnp, numero_val_significativi,grado);
					
					hipMemcpy(risultati+(i*col_ospitabili*Nr_vet_elem), Ris, cicle_dim, hipMemcpyDeviceToHost);
					
					contatore = contatore + col_ospitabili;
				}
				
				resto = Nr_righe - contatore;
				
				if (resto > 0)
				{
					for (int kk = 0; kk < resto; kk++)
					{
						indici[kk] = contatore + kk;
					}
					
					hipMemcpy(ind, indici, indici_size, hipMemcpyHostToDevice);
					
					Kernel_polimoniale<<< blockGridRows, threadBlockRows>>>(Vd, Ris, Nr_vet_elem, Nr_vet_comp, resto, ind, Vp, Vnp, numero_val_significativi,grado);
					
					hipMemcpy(risultati+(numero_cicli)*(col_ospitabili*Nr_vet_elem), Ris, resto * Nr_vet_elem * sizeof(float), hipMemcpyDeviceToHost);
					
				}
				
				break;
			}
			
			case(3):{
				//cout<<"Kernel gaussiano\n";
				
				/**
				* Sigma della funzione gaussiana.
				**/
				float sigma = atof(argv[5]);
				sigma = (1.0/(2.0*sigma*sigma));
				
				//calcolo norme.
				Kernel_norme<<< blockGridRowsn, 256 >>>(Vd, Nd, Vp, Vnp, Nr_vet_elem, Nr_vet_comp, numero_val_significativi);
				
				//calcolo kernel
				for(int i = 0; i < numero_cicli; i++)
				{
					for (int kk = 0; kk < col_ospitabili; kk++)
					{
						indici[kk] = contatore + kk;
					}
					
					hipMemcpy(ind, indici, indici_size, hipMemcpyHostToDevice);
					
					//Kernel_gaus(float *Vd, float *Ris, float *Nd, int N, int C, int dim_indici, int *ind, float sigma, float *Vp, float *Vnp)
					Kernel_gaus<<< blockGridRows, threadBlockRows>>>(Vd, Ris, Nd, Nr_vet_elem, Nr_vet_comp, col_ospitabili, ind, sigma, Vp, Vnp, numero_val_significativi);
					
					hipMemcpy(risultati+(i*col_ospitabili*Nr_vet_elem), Ris, cicle_dim, hipMemcpyDeviceToHost);
					
					contatore = contatore + col_ospitabili;
				}
				
				resto = Nr_righe - contatore;
				
				if (resto > 0)
				{
					for (int kk = 0; kk < resto; kk++)
					{
						indici[kk] = contatore + kk;
					}
					
					hipMemcpy(ind, indici, indici_size, hipMemcpyHostToDevice);
					
					Kernel_gaus<<< blockGridRows, threadBlockRows>>>(Vd, Ris, Nd, Nr_vet_elem, Nr_vet_comp, resto, ind, sigma, Vp, Vnp, numero_val_significativi);

					
					hipMemcpy(risultati+(numero_cicli)*(col_ospitabili*Nr_vet_elem), Ris, resto * Nr_vet_elem * sizeof(float), hipMemcpyDeviceToHost);
					
				}
				
				break;
			}
			
			default:
			{
			cout<<"Scelta non valida.\n";
			cout<<"4° argomento non esistente.\n";
			cout<<"1 = kernel lineare\t2 = kernel polimoniale\t 3 = kernel gaussiano\n";
			break;
			}
			
			
			
	}
	
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	hipEventElapsedTime(&elapsedTime, start,stop);
	cout<<"Tempo totale:\t"<<elapsedTime/1000<<" secondi\n";
	
	/*for (int i = 0; i <  Nr_vet_elem*Nr_righe; i++)
	{
	  cout<<risultati[i]<<endl;
	}*/
	
	free(vettori);
	free(vett_norme);
	free(vettore_posizioni);
	free(vett_numero_posizioni);
	free(indici);
	free(risultati);
	
	hipFree(Vd);
	hipFree(Vp);
	hipFree(Vnp);
	hipFree(Nd);
	hipFree(ind);
	hipFree(Ris);
	
	return 0;
}
